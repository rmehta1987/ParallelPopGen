#include "hip/hip_runtime.h"
/*
 * go_fish_impl.cu
 *
 *      Author: David Lawrie
 *      implementation of non-template, non-inline functions for GO Fish simulation
 */

#include "../_internal/go_fish_impl.cuh"

//!\cond
namespace go_fish_details{

/*
 *  CUB scan (sum) phenomenon: float errors in the mse_integral can accumulate differently each run
 *  e.g.
 *  GO_Fish::const_parameter mutation(pow(10.f,-9)); //per-site mutation rate
	GO_Fish::const_parameter inbreeding(1.f); //constant inbreeding
	GO_Fish::const_demography demography(pow(10.f,5)*(1+inbreeding(0,0))); //number of individuals in population, set to maintain consistent effective number of chromosomes
	GO_Fish::const_equal_migration migration(0.f,a.sim_input_constants.num_populations); //constant migration rate
	float gamma = -5; //effective selection
	GO_Fish::const_selection selection(gamma/(2*demography(0,0))); //constant selection coefficient
	GO_Fish::const_parameter dominance(0.f); //constant allele dominance
 *  a.sim_input_constants.compact_interval = 20;
    a.sim_input_constants.num_generations = pow(10.f,3);
    a.sim_input_constants.num_sites = 20*2*pow(10.f,7);
    a.sim_input_constants.seed1 = 0xbeeff00d + 2*14; //random number seeds
    a.sim_input_constants.seed2 = 0xdecafbad - 2*14;

    one solution is to round the results in reverse array
    another is to ignore (currently implemented)
    another is to switch back to using doubles ... at least when summing up the mse_integral (difference in speed was slight)
 */

__global__ void reverse_array(float * array, const int N){
	int myID = blockIdx.x*blockDim.x + threadIdx.x;
	for(int id = myID; id < N/2; id += blockDim.x*gridDim.x){
		float temp = array[N - id - 1];
		array[N - id - 1] = array[id];
		//float temp = roundf(10000*array[N - id - 1])/10000.f;
		//array[N - id - 1] = roundf(10000*array[id])/10000.f;
		array[id] = temp;
	}
}

__global__ void initialize_mse_mutation_array(float * mutations_freq, const int * freq_index, const int * scan_index, const int offset, const int Nchrom, const int population, const int num_populations, const int array_Length){
	int myIDy = blockIdx.y*blockDim.y + threadIdx.y;
	for(int idy = myIDy; idy < (Nchrom-1); idy+= blockDim.y*gridDim.y){
		int myIDx = blockIdx.x*blockDim.x + threadIdx.x;
		int start = scan_index[offset+idy];
		int num_mutations = freq_index[offset+idy];
		float freq = (idy+1.f)/Nchrom;
		for(int idx = myIDx; idx < num_mutations; idx+= blockDim.x*gridDim.x){
			for(int pop = 0; pop < num_populations; pop++){ mutations_freq[pop*array_Length + start + idx] = 0; }
			mutations_freq[population*array_Length + start + idx] = freq;
		}
	}
}

__global__ void mse_set_mutID(int4 * mutations_ID, const float * const mutations_freq, const int mutations_Index, const int num_populations, const int array_Length, const bool preserve_mutations){
	int myID = blockIdx.x*blockDim.x + threadIdx.x;
	for(int id = myID; id < mutations_Index; id+= blockDim.x*gridDim.x){
		for(int pop = 0; pop < num_populations; pop++){
			if(mutations_freq[pop*array_Length+id] > 0){
				if(!preserve_mutations){ mutations_ID[id] = make_int4(0,pop,(id+1),0); }
				else{ mutations_ID[id] = make_int4(0,pop,-1*(id+1),0); } //age: eventually will replace where mutations have age <= 0 (age before sim start)//threadID//population//to ensure that ID is non-zero, so that preservation flag can be a -ID
				break; //assumes mutations are only in one population at start
			}
		}
	}
}

/*__global__ void print_Device_array_uint(unsigned int * array, int num){

	for(int i = 0; i < num; i++){
		//if(i%1000 == 0){ printf("\n"); }
		printf("%d: %d\t",i,array[i]);
	}
}

__global__ void sum_Device_array_bit(unsigned int * array, int num){
//	int sum = 0;
	for(int i = 0; i < num; i++){
		//if(i%1000 == 0){ printf("\n"); }
		unsigned int n = array[i];
		while (n) {
		    if (n & 1)
		    	sum+=1;
		    n >>= 1;
		}
		printf("%d\t",__popc(array[i]));
	}
}

__global__ void sum_Device_array_uint(unsigned int * array, int num){
	int j = 0;
	for(int i = 0; i < num; i++){
		j += array[i];
	}
	printf("%d",j);
}

__global__ void sum_Device_array_float(float * array, int start, int end){
	double j = 0;
	for(int i = start; i < end; i++){
		j += array[i];
	}
	printf("%lf\n",j);
}

__global__ void compareDevicearray(int * array1, int * array2, int array_length){
	int myID =  blockIdx.x*blockDim.x + threadIdx.x;
	for(int id = myID; id < array_length; id+= blockDim.x*gridDim.x){
		if(array1[id] != array2[id]){ printf("%d,%d,%d\t",id,array1[id],array2[id]); }
	}
}

__global__ void copyDevicearray(int * array1, int * array2, int array_length){
	int myID =  blockIdx.x*blockDim.x + threadIdx.x;
	for(int id = myID; id < array_length; id+= blockDim.x*gridDim.x){ array1[id] = array2[id]; }
}

__global__ void compareDevicearray(float * array1, float * array2, int array_length){
	int myID =  blockIdx.x*blockDim.x + threadIdx.x;
	for(int id = myID; id < array_length; id+= blockDim.x*gridDim.x){
		if(array1[id] != array2[id]){ printf("%d,%f,%f\t",id,array1[id],array2[id]); return; }
	}
}

__global__ void copyDevicearray(float * array1, float * array2, int array_length){
	int myID =  blockIdx.x*blockDim.x + threadIdx.x;
	for(int id = myID; id < array_length; id+= blockDim.x*gridDim.x){ array1[id] = array2[id]; }
}

__global__ void print_Device_array_float(float * array, int num){
	printf("%5.10e\n",array[num]);
}*/

__global__ void add_new_mutations(float * mutations_freq, int4 * mutations_ID, const int prev_mutations_Index, const int new_mutations_Index, const int array_Length, float freq, const int population, const int num_populations, const int generation){
	int myID =  blockIdx.x*blockDim.x + threadIdx.x;
	for(int id = myID; (id < (new_mutations_Index-prev_mutations_Index)) && ((id + prev_mutations_Index) < array_Length); id+= blockDim.x*gridDim.x){
		for(int pop = 0; pop < num_populations; pop++){ mutations_freq[(pop*array_Length+prev_mutations_Index+id)] = 0; }
		mutations_freq[(population*array_Length+prev_mutations_Index+id)] = freq;
		mutations_ID[(prev_mutations_Index+id)] = make_int4(generation,population,(id+1),0); //to ensure that ID is non-zero, so that preservation flag can be a -ID
	}
}

__global__ void scatter_arrays(float * new_mutations_freq, int4 * new_mutations_ID, const float * const mutations_freq, const int4 * const mutations_ID, const unsigned int * const flag, const unsigned int * const scan_Index, const int padded_mut_Index, const int new_array_Length, const int old_array_Length, const bool preserve_mutations, const int warp_size){
//adapted from https://www.csuohio.edu/engineering/sites/csuohio.edu.engineering/files/Research_Day_2015_EECS_Poster_14.pdf
	int myID =  blockIdx.x*blockDim.x + threadIdx.x;
	int population = blockIdx.y;

	for(int id = myID; id < (padded_mut_Index >> 5); id+= blockDim.x*gridDim.x){
		int lnID = threadIdx.x % warp_size;
		int warpID = id >> 5;

		unsigned int predmask;
		unsigned int cnt;

		predmask = flag[(warpID<<5)+lnID];
		cnt = __popc(predmask);

		//parallel prefix sum
#pragma unroll
		for(int offset = 1; offset < 32; offset<<=1){
			unsigned int n = __shfl_up(cnt, offset);
			if(lnID >= offset) cnt += n;
		}

		unsigned int global_index = 0;
		if(warpID > 0) global_index = scan_Index[warpID - 1];

		for(int i = 0; i < 32; i++){
			unsigned int mask = __shfl(predmask, i); //broadcast from thread i
			unsigned int sub_group_index = 0;
			if(i > 0) sub_group_index = __shfl(cnt, i-1);
			if(mask & (1 << lnID)){
				int write = global_index + sub_group_index + __popc(mask & ((1 << lnID) - 1));
				int read = (warpID<<10)+(i<<5)+lnID;
				new_mutations_freq[population*new_array_Length + write] = mutations_freq[population*old_array_Length+read];
				if(population == 0){
					if(preserve_mutations){
						int4 ID = mutations_ID[read];
						new_mutations_ID[write] = make_int4(ID.x,ID.y,-1*abs(ID.z),ID.w);
					}else{ new_mutations_ID[write] = mutations_ID[read]; }
				}
			}
		}
	}
}

__global__ void preserve_prev_run_mutations(int4 * mutations_ID, const int mutations_Index){
	int myID =  blockIdx.x*blockDim.x + threadIdx.x;
	for(int id = myID; id < mutations_Index; id+= blockDim.x*gridDim.x){ mutations_ID[id].z = -1*abs(mutations_ID[id].z); } //preservation flag is a -ID, use of absolute value is to ensure that if ID is already
}

} /* ----- end namespace go_fish_details ----- */
//!\endcond
