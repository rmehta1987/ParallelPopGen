#include "hip/hip_runtime.h"
/*
 * run.cu
 *
 *      Author: Rahul Mehta
 */

#include "go_fish.cuh"
#include "spectrum.h"
#include <vector>
#include <fstream>
#include <iterator>

/*
This folllows the simple zig-zag demographic model of schiffels-dubin for a single population:
https://github.com/popsim-consortium/demes-python/blob/main/examples/zigzag.yaml


description: A single population model with epochs of exponential growth and decay.
doi:
  - https://doi.org/10.1038/ng.3015
time_units: generations
demes:
  - name: generic
    description: All epochs wrapped into the same population, so that epoch intervals
      do not overlap, and they tile the entire existence of the population (all time,
      in this case).

epochs:
   0 - end_time: 34133.31
      start_size: 7156
   1 - end_time: 8533.33
      end_size: 71560
   2 - end_time: 2133.33
      end_size: 7156
   3 - end_time: 533.33
      end_size: 71560
   4 - end_time: 133.33
      end_size: 7156
   5 - end_time: 33.333
      end_size: 71560
   6 - end_time: 0
      end_size: 71560

I'm pretty sure these are going backwards (coalescent) as in the simulation starts 34133 generations ago and ends 0 generations ago (modern day), for forward simulations you need to invert the order of the generations e.g.:

epochs:
   0 - end_time: 0
      end_size: 7156
   1 - end_time: 25,599.98 (= 34133.31 - 8533.33 + 0)
      end_size: 71560
   2 - end_time: 31,999.98 (= 8533.33 - 2133.33 + 25,599.98)
      end_size: 7156
   3 - end_time: 33,599.98
      end_size: 71560
   4 - end_time: 33,999.98
      end_size: 7156
   5 - end_time: 34,099.98
      end_size: 71560
   6 - end_time: 34133.31
      end_size: 71560
 */

void run_validation_test(float mut_rate, float sel_coef, int num_samples){

    typedef Sim_Model::migration_constant_equal mig_const; // no migration
	

	GO_Fish::allele_trajectories b;
	b.sim_input_constants.num_populations = 1; 							//number of populations

    b.sim_input_constants.num_generations = 34150;
    b.sim_input_constants.num_sites = 36.f*pow(10.f,6);	 // Should be 36 Megabase pairs 
    b.sim_input_constants.compact_interval = 10;
    // Mutation and dominance parameters TODO Change dominance paramater to that of stabalizing selection

	Sim_Model::F_mu_h_constant codominant(0.5f); 						//dominance (co-dominant)
	Sim_Model::F_mu_h_constant outbred(0.f); 							//inbreeding (outbred)

	//Sim_Model::F_mu_h_constant mutation((float) mut_rate / (b.num_sites())); 	//per-site mutation rate 10^-9
    Sim_Model::F_mu_h_constant mutation(9.61111f*pow(10.f,-10)); 				//per-site 
    //std::cout << "mutation rate: " << 9.61f*pow(10.f,-10) << std::endl;
    

    // Demographic model
   	int N_ind = 7156;					//initial number of individuals in population
	Sim_Model::demography_constant demography_model(N_ind);	
 
    // Migration parameters, no--migration
    mig_const mig_model;

    // Selection parameters
	Sim_Model::selection_constant weak_del((float) sel_coef);

    // SFS parameters
	int sample_size = num_samples;										//number of samples in SFS
	int num_iter = 50;													//number of iterations
    Spectrum::SFS my_spectra;

    hipEvent_t start, stop;											//CUDA timing functions
    float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float avg_num_mutations = 0;
	float avg_num_mutations_sim = 0;
	std::vector<std::vector<float> > results(num_iter); 				//storage for SFS results
	for(int j = 0; j < num_iter; j++){ results[j].reserve(sample_size); }

	for(int j = 0; j < num_iter; j++){
		if(j == num_iter/2){ hipEventRecord(start, 0); } 				//use 2nd half of the simulations to time simulation runs + SFS creation

		b.sim_input_constants.seed1 = 0xbeeff00d + 2*j; 				//random number seeds
		b.sim_input_constants.seed2 = 0xdecafbad - 2*j;
		GO_Fish::run_sim(b, mutation, demography_model, mig_model, weak_del, outbred, codominant, Sim_Model::bool_off(), Sim_Model::bool_off());
		Spectrum::site_frequency_spectrum(my_spectra,b,0,0,sample_size);

		avg_num_mutations += ((float)my_spectra.num_mutations)/num_iter;
		avg_num_mutations_sim += b.maximal_num_mutations()/num_iter;
        
		for(int i = 0; i < sample_size; i++){ results[j][i] = my_spectra.frequency_spectrum[i]; }
	}

	elapsedTime = 0;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//output SFS simulation results
	std::cout<<"SFS :"<<std::endl<< "allele count\tavg# mutations\tstandard dev\tcoeff of variation (aka relative standard deviation)"<< std::endl;
	for(int i = 1; i < sample_size; i++){
		double avg = 0;
		double std = 0;
		float num_mutations;
		for(int j = 0; j < num_iter; j++){ num_mutations = b.num_sites() - results[j][0]; avg += results[j][i]/(num_iter*num_mutations); }
		for(int j = 0; j < num_iter; j++){ num_mutations = b.num_sites() - results[j][0]; std += 1.0/(num_iter-1)*pow(results[j][i]/num_mutations-avg,2); }
		std = sqrt(std);
		std::cout<<i<<"\t"<<avg<<"\t"<<std<<"\t"<<(std/avg)<<std::endl;
	}

    std::ofstream output_file("./out_sfs.txt");
    for(int i = 0; i < sample_size; i++)
    { 
        output_file << my_spectra.frequency_spectrum[i] << "\n"; 
    }
    

	std::cout<<"\nnumber of sites in simulation: "<< b.num_sites() <<"\ncompact interval: "<< b.last_run_constants().compact_interval;
	std::cout<<"\naverage number of mutations in simulation: "<<avg_num_mutations_sim<<"\naverage number of mutations in SFS: "<<avg_num_mutations<<"\ntime elapsed (ms): "<< 2*elapsedTime/num_iter<<std::endl;
}

////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////

int main(int argc, char **argv) 

{ 
     // this is the mutation rate scaled with respect to number of sites, mutation_rate*(number of sites)
    float mut_rate = 0.3426;    
    // this is a point selection coefficient the selection coefficient will remain the same for the population, this is the un-scaled selection coefficient
    float PointSel = -.005; 
    int num_samples = 100;    

    // Number of samples for to generate for the site-frequency spectrum (SFS

    // Eventually this will read in a demographic history file for easier command line use instead of having to re-compile for every new demography <- possible but will still require a compilation step as Functors (functions passed as templates) need to be known at compile time (a requirement of GPUs), I have not yet added the ability to do this to the library, technically there are other libraries that will allow this, but I haven't merged them with my API to make it easy. It's on my TODO list


    if (argc != 4) // 3 Total parameters, [executable, scaled mutation rate, unscaled selection coefficient, num_samples]
    {
        fprintf(stderr, "Error: The number of arguments given in the command line is not correct. In this version you need to pass in a selection cofficient and unscaled mutation rate, format is: ./GOFish scaled_mutation_rate unscaled_selection coefficient num_samples \n");
        //exit(8);
        std::cout << "Using default values" << std::endl;
    }
    else{

        mut_rate = atof(argv[1]);
        PointSel = atof(argv[2]);
        num_samples = atoi(argv[3]);
    }

    std::cout<<"Scaled Mutation Rate: " << mut_rate << std::endl;
    std::cout<<"Inscaled Point Selection: " << PointSel << std::endl;
    std::cout<<"Number of samples to generate SFS: " << num_samples << std::endl;



    std::cout<<"Running simulations" << std::endl;

    run_validation_test(mut_rate, PointSel, num_samples); 
    
    }
